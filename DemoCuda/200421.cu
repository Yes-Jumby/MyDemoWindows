#include "hip/hip_runtime.h"
#include <iostream>
#include "time.h"
#include "cv.hpp"
#include "cudaarithm.hpp"
#include "cudafilters.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
using namespace cv;
using namespace std;

__global__ void Continuation(cuda::PtrStepSz<float1> input, cuda::PtrStepSz<float1> output,int size,uint cols,uint rows)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	while (idx < size)
	{
		int distorted_x = idx%cols;//width
		int distorted_y = idx / cols;//height
		output(distorted_y, distorted_x) = input(distorted_y, distorted_x);
		output(distorted_y, 2 * cols - 1 - distorted_x) = input(distorted_y, distorted_x);
		output(2 * rows - 1 - distorted_y, distorted_x) = input(distorted_y, distorted_x);
		output(2 * rows - 1 - distorted_y, 2 * cols - 1 - distorted_x) = input(distorted_y, distorted_x);
		idx += stride;
	}
}
int main()
{
	clock_t a; 
	int size_ = 1000;
	int testtimes = 10;
	for (int aaa = 1; aaa < 11; aaa++)
	{
		size_ = aaa * 500;
		cout << "Mat Size" << size_ << endl;
		cv::Mat Rol = cv::Mat::ones(size_, size_, CV_32F);
		Rol.at<float>(0, 0) = 1;
		Rol.at<float>(0, 1) = 2;
		Rol.at<float>(1, 0) = 3;
		Rol.at<float>(1, 1) = 4;
		cv::Mat HresultImage;
		cv::Mat C;
		cv::Mat VresultImage;
		a = clock();
		for (int mytimes = 0; mytimes < testtimes; mytimes++)
		{
			cv::flip(Rol, HresultImage, 1);
			cv::hconcat(Rol, HresultImage, HresultImage);
			cv::flip(HresultImage, VresultImage, 0);
			cv::vconcat(HresultImage, VresultImage, C);
		}
		cout << "Opencv cpu�⺯��" << clock() - a << endl;
		a = clock();
		cv::Mat RolP = cv::Mat(cv::Size(2 * Rol.cols, 2 * Rol.rows), Rol.type(), cv::Scalar(0));
		for (int mytimes = 0; mytimes < testtimes; mytimes++)
		{
			for (int i = 0; i < Rol.cols; ++i)
			{
				for (int j = 0; j < Rol.rows; ++j)
				{
					RolP.at<float>(j, i) = Rol.at<float>(j, i);
					RolP.at<float>(j, 2 * Rol.cols - 1 - i) = Rol.at<float>(j, i);
					RolP.at<float>(2 * Rol.rows - 1 - j, i) = Rol.at<float>(j, i);
					RolP.at<float>(2 * Rol.rows - 1 - j, 2 * Rol.cols - 1 - i) = Rol.at<float>(j, i);
				}
			}
		}
		cout << "��дCPU����" << clock() - a << endl;

		a = clock();
		cuda::GpuMat GRol;
		GRol.upload(Rol);
		cuda::GpuMat GRolP(cv::Size(2 * Rol.cols, 2 * Rol.rows), Rol.type(), Scalar(0, 0, 0));
		int size = Rol.cols*Rol.rows;
		for (int mytimes = 0; mytimes < testtimes; mytimes++)
		{
			Continuation << <(size + 1023) / 1024, 1024 >> > (GRol, GRolP, size, Rol.cols, Rol.rows);
			hipDeviceSynchronize();
		}
		cout << "��дGPU����" << clock() - a << endl;
		cout << endl << endl;
		Mat temp;
		GRolP.download(temp);
	}
	
	
	
	return 0;
}