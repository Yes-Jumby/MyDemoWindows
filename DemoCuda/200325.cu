// CUDA runtime �� + CUBLAS ��
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <iostream>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
using namespace std;
using namespace cv;
// ������Ծ����ά��
int const A_ROW = 3;
int const A_COL = 1;
int const B_ROW = 1;
int const B_COL = 3;

int main11222221()
{

	Mat X = Mat::ones(2, 2448, CV_32F);
	for (int i = 0; i < 2448; i++)
	{
		X.at<float>(0, i)=i+1;
	}
	X = X.t();
	Mat COFFFFF = X*((X.t()*X).inv())*X.t();//
	// ����״̬����
	hipblasStatus_t status;
	 //���ڴ��п��ٿռ�
	float *h_A, *h_B, *h_C;    //�洢���Դ��еľ���
	h_A = (float*)malloc(sizeof(float)*B_ROW*B_COL);
	h_B = (float*)malloc(sizeof(float)*B_ROW*B_COL);
	h_C = (float*)malloc(sizeof(float)*A_ROW*B_COL);

	//0.414321288604656	0.325286165381830	0.324780957273329	0.325031346860598	0.325142287794897
	// Ϊ����������Ԫ�ظ��� 0-10 ��Χ�ڵ������
	for (int i = 0; i<A_ROW*A_COL; i++) {
		h_A[i] = (float)(rand() % 10 + 1);
	}
	for (int i = 0; i<B_ROW*B_COL; i++) {
		h_B[i] = (float)(rand() % 10 + 1);
	}
	// ��ӡ�����Եľ���
	cout << "���� A :" << endl;
	for (int i = 0; i<A_ROW*A_COL; i++) {
		cout << h_A[i] << " ";
		if ((i + 1) % A_COL == 0) cout << endl;
	}
	cout << endl;
	cout << "���� B :" << endl;
	for (int i = 0; i<B_ROW*B_COL; i++) {
		cout << h_B[i] << " ";
		if ((i + 1) % B_COL == 0) cout << endl;
	}
	cout << endl;

	float *d_A, *d_B, *d_C;    //�洢���Դ��еľ���
	hipMalloc((void**)&d_A, sizeof(float)*A_ROW*A_COL); //���Դ��п��ٿռ�
	hipMalloc((void**)&d_B, sizeof(float)*B_ROW*B_COL);
	hipMalloc((void**)&d_C, sizeof(float)*A_ROW*B_COL);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipMemcpy(d_A, h_A, sizeof(float)*A_ROW*A_COL, hipMemcpyHostToDevice); //���ݴ��ڴ濽�����Դ�
	hipMemcpy(d_B, h_B, sizeof(float)*B_ROW*B_COL, hipMemcpyHostToDevice);

	float a = 1, b = 0;
	hipblasSgemm(
		handle,
		HIPBLAS_OP_T,   //����A�����Բ�����ת�ã���������
		HIPBLAS_OP_T,   //����B�����Բ�����ת�ã���������
		A_ROW,          //����A��C������2448
		B_COL,          //����B��C������2448
		A_COL,          //A��������B���������˴�Ҳ��ΪB_ROW,һ����2448
		&a,             //alpha��ֵ
		d_A,            //�����ΪA
		A_COL,          //A��leading dimension����ʱѡ��ת�ã��������ȣ���leading dimensionΪA������
		d_B,            //�Ҿ���ΪB
		B_COL,          //B��leading dimension����ʱѡ��ת�ã��������ȣ���leading dimensionΪB������
		&b,             //beta��ֵ
		d_C,            //�������C
		A_ROW           //C��leading dimension��C����һ���������ȣ���leading dimensionΪC������
	);
	//��ʱ�õ��Ľ������C=AB,������C�ǰ������ȣ��ʴ�ʱ�õ���CӦ������ȷ�����ת��
	std::cout << "��������ת�� ( (A*B)��ת�� )��" << std::endl;


	hipMemcpy(h_C, d_C, sizeof(float)*A_ROW*B_COL, hipMemcpyDeviceToHost);
	for (int i = 0; i<A_ROW*B_COL; ++i) {
		std::cout << h_C[i] << " ";
		if ((i + 1) % B_COL == 0) std::cout << std::endl;
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}