#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <malloc.h>
#include <string.h>
const int size = 10 * 1024 * 1024;
const int N = 1024 * 1024;
const int FULL_DATA_SIZE = 2 * N;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256;

//����cudamalloc����ѭ��100�ε�����ʱ��
float cuda_malloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (int *)malloc(10 * 1024 * 1024 * sizeof(int));
	printf("%p", a);
	if (a == NULL)
	{
		printf("host memory malloc fail!\n");
	}

	hipError_t cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(*dev_a));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!\n");
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++)
	{
		if (up)
		{
			cudaStatus = hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy Host to Device failed!\n");
				return -1;
			}
		}
		else
		{
			cudaStatus = hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy Device to Host failed!\n");
				return -1;
			}
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	free(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;

}

//����hipHostAlloc����ѭ��100�ε�����ʱ��
float cuda_host_alloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus = hipHostAlloc((void **)&a, size * sizeof(*a), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
		printf("host alloc fail!\n");
		return -1;
	}

	cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(*dev_a));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!\n");
		return -1;
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++)
	{
		if (up)
		{
			cudaStatus = hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy Host to Device failed!\n");
				return -1;
			}
		}
		else
		{
			cudaStatus = hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy Device to Host failed!\n");
				return -1;
			}
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipHostFree(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;

}

//�˺���������һ�����д�ļӷ�
__global__ void kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N)
	{
		for (int i = 0; i < 2000; i++)
		{
			int idx1 = (idx + 1) % GPUTHREADNUM;
			int idx2 = (idx + 2) % GPUTHREADNUM;
			float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
			float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
			c[idx] = (as + bs) / 2;
		}
		
	}
}

//�˺���������һ�����д�ļӷ�
__global__ void kernel1(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N)
	{
		a[idx] = 33;
		b[idx] = 33;
		c[idx] = 33;
	}
}
//�˺���������һ�����д�ļӷ�
__global__ void kernel2(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 10)
	{
		c[idx] = 10;
	}
}

void mytest()
{
	//����hipHostAlloc���������̶��ڴ�
	int *host_a;
	int *dev_a;
	hipError_t cudaStatus;
	cudaStatus = hipHostAlloc((void **)&host_a, 10 * sizeof(int), hipHostMallocMapped| hipHostMallocWriteCombined);
	hipHostGetDevicePointer((void**)&dev_a, (void*)host_a, hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
		printf("hipHostAlloc host_a failed!\n");
	}
	//������������
	for (int i = 0; i < 10; i++)
	{
		host_a[i] = i;
		printf("Time taken : %d ms\n", host_a[i]);
	}

	
	kernel2 << <N / GPUBLOCKNUM, GPUTHREADNUM >> >(dev_a, dev_a, dev_a);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("hipDeviceSynchronize failed!\n");
	}
	printf("Time taken : %d ms\n", host_a[0]);
	printf("Time taken : %d ms\n", host_a[1]);
	printf("Time taken : %d ms\n", host_a[2]);
	printf("Time taken : %d ms\n", host_a[3]);
	printf("Time taken : %d ms\n", host_a[4]);
	int a[10] = { 0 };
	hipMemcpy(a, host_a, 10 * sizeof(int), hipMemcpyHostToHost);
	hipMemcpy(a, dev_a, 10 * sizeof(int), hipMemcpyDeviceToHost);
	//memcpy(a, host_a,10* sizeof(int));
	printf("Time taken : %d ms\n", a[0]);
	printf("Time taken : %d ms\n", a[1]);
	printf("Time taken : %d ms\n", a[2]);
	printf("Time taken : %d ms\n", a[3]);
	printf("Time taken : %d ms\n", a[4]);
	int b[10] = { 100 };
	hipMemcpy(host_a, b, 10 * sizeof(int), hipMemcpyHostToHost);
	printf("Time taken : %d ms\n", host_a[0]);
	printf("Time taken : %d ms\n", host_a[1]);
	printf("Time taken : %d ms\n", host_a[2]);
	printf("Time taken : %d ms\n", host_a[3]);
	printf("Time taken : %d ms\n", host_a[4]);
	hipMemcpy(a, dev_a, 10 * sizeof(int), hipMemcpyDeviceToHost);

	return;
}
void singlestream()
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not handle overlaps, so no speed up from streams.\n");
		return ;
	}

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipStream_t stream;
	hipStreamCreate(&stream);
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void **)&dev_a, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
	printf("hipMalloc dev_a failed!\n");
	}
	cudaStatus = hipMalloc((void **)&dev_b, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
	printf("hipMalloc dev_b failed!\n");
	}
	cudaStatus = hipMalloc((void **)&dev_c, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
	printf("hipMalloc dev_c failed!\n");
	}
	cudaStatus = hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
	printf("hipHostAlloc host_a failed!\n");
	}
	cudaStatus = hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
	printf("hipHostAlloc host_b failed!\n");
	}
	cudaStatus = hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
	printf("hipHostAlloc host_c failed!\n");
	}
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
	host_a[i] = i - N;
	host_b[i] = i;
	}
	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		cudaStatus = hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		if (cudaStatus != hipSuccess)
		{
			printf("hipMemcpyAsync a failed!\n");
		}
		cudaStatus = hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		if (cudaStatus != hipSuccess)
		{
			printf("hipMemcpyAsync b failed!\n");
		}
		kernel << <N / GPUBLOCKNUM, GPUTHREADNUM, 0, stream >> >(dev_a, dev_b, dev_c);
		cudaStatus = hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
		if (cudaStatus != hipSuccess)
		{
			printf("hipMemcpyAsync c failed!\n");
		}
	}
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess)
	{
	printf("hipStreamSynchronize failed!\n");
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Singlestream Time taken : %3.1f ms\n", elapsedTime);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);//165ms
	hipFree(dev_c);
	hipStreamDestroy(stream);
}
//���ú˺�����ⵥ����������ʱ���������������ʱ�䣬�Ƚ���Ч��
int mutistream()
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not handle overlaps, so no speed up from streams.\n");
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	int *host_a, *host_b, *host_c;
	int *dev0_a, *dev0_b, *dev0_c;
	int *dev1_a, *dev1_b, *dev1_c;

	//����hipMalloc��������GPU�ڴ�
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void **)&dev0_a, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev0_a failed!\n");
	}

	cudaStatus = hipMalloc((void **)&dev0_b, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev0_b failed!\n");
	}

	cudaStatus = hipMalloc((void **)&dev0_c, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev0_c failed!\n");
	}

	cudaStatus = hipMalloc((void **)&dev1_a, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev1_a failed!\n");
	}

	cudaStatus = hipMalloc((void **)&dev1_b, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev1_b failed!\n");
	}

	cudaStatus = hipMalloc((void **)&dev1_c, N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc dev1_c failed!\n");
	}

	//����hipHostAlloc���������̶��ڴ�
	cudaStatus = hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
		printf("hipHostAlloc host_a failed!\n");
	}

	cudaStatus = hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
		printf("hipHostAlloc host_b failed!\n");
	}

	cudaStatus = hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	if (cudaStatus != hipSuccess)
	{
		printf("hipHostAlloc host_c failed!\n");
	}

	//������������
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = i - N;
		host_b[i] = i;
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N * 2)
	{
		cudaStatus = hipMemcpyAsync(dev0_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync0 a failed!\n");
		}

		cudaStatus = hipMemcpyAsync(dev0_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync0 b failed!\n");
		}

		kernel << <N / GPUBLOCKNUM, GPUTHREADNUM, 0, stream0 >> >(dev0_a, dev0_b, dev0_c);

		cudaStatus = hipMemcpyAsync(dev1_a, host_a + N + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync1 a failed!\n");
		}

		cudaStatus = hipMemcpyAsync(dev1_b, host_b + N + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync1 b failed!\n");
		}

		kernel << <N / GPUBLOCKNUM, GPUTHREADNUM, 0, stream1 >> >(dev1_a, dev1_b, dev1_c);

		cudaStatus = hipMemcpyAsync(host_c + i, dev0_c, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync0 c failed!\n");
		}

		cudaStatus = hipMemcpyAsync(host_c + N + i, dev1_c, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
		if (cudaStatus != hipSuccess)
		{
			printf("cudaMemcpyAsync1 c failed!\n");
		}
	}
	
	cudaStatus = hipStreamSynchronize(stream0);
	if (cudaStatus != hipSuccess)
	{
		printf("cudaStreamSynchronize0 failed!\n");
	}

	cudaStatus = hipStreamSynchronize(stream1);
	if (cudaStatus != hipSuccess)
	{
		printf("cudaStreamSynchronize1 failed!\n");
	}
	kernel1 << <N / GPUBLOCKNUM, GPUTHREADNUM >> >(dev1_a, dev1_b, dev1_c);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("hipDeviceSynchronize failed!\n");
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Mutistream Time taken : %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev0_a);
	hipFree(dev0_b);
	hipFree(dev0_c);
	hipFree(dev1_a);
	hipFree(dev1_b);
	hipFree(dev1_c);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);//162

	hipProfilerStop();
	
	return 0;

}

int main0317()
{
	singlestream();
	mutistream();
	return 0;
}

/* ����������cuda_malloc_test������cuda_host_alloc_test�������й̶��ڴ�Ĳ���*/
int main1(int argc, char ** argv)
{
	float elapsedTime;
	float MB = (float)100 * size * sizeof(int) / 1024 / 1024;
	elapsedTime = cuda_malloc_test(size, true);
	printf("Time using hipMalloc: %3.1f ms.", elapsedTime);
	printf("\tMB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_malloc_test(size, false);
	printf("Time using hipMalloc: %3.1f ms.", elapsedTime);
	printf("\tMB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_host_alloc_test(size, true);
	printf("Time using hipHostAlloc: %3.1f ms.", elapsedTime);
	printf("\tMB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_host_alloc_test(size, false);
	printf("Time using hipHostAlloc: %3.1f ms.", elapsedTime);
	printf("\tMB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	return 0;
}
