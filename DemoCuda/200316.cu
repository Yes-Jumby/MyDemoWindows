#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#define max(a,b) (a>b?a:b)
#include <opencv2/opencv.hpp>
using namespace std;
using namespace cv;
texture<float> t_input;
__global__ void MaxPool2d(const int height, const int pooled_height, float* top_data)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int dx = gridDim.x;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dtx = blockDim.x;
	int dty = blockDim.y;
	float s = -10000.0;
	float a1, a2, a3, a4, a12, a34;
	int index2 = y*dx*dtx*dty + x*dtx*dty + ty*dtx + tx;
	int index = y*dx*height*height + x*height*height + ty*pooled_height*height + tx*pooled_height;
	int index3 = 0;
	bool b1 = 1 + 2 * ty == height;
	bool b2 = 1 + 2 * tx == height;

	if (b1&&b2)
	{
		a1 = tex1Dfetch(t_input, index);
		s = max(a1, s);
	}
	if (!b2)
	{
		a1 = tex1Dfetch(t_input, index);
		a2 = tex1Dfetch(t_input, index + 1);
		a12 = max(a1, a2);
		s = max(a12, s);
		index3 = height + 1;
	}
	if (!b1)
	{
		a3 = tex1Dfetch(t_input, index + index3);
		a4 = tex1Dfetch(t_input, index + height);
		a34 = max(a3, a4);
		s = max(a34, s);
	}
	*(top_data + index2) = s;
}
__global__ void GetThres(
	unsigned char				*IMG,
	unsigned int		cloud_size,
	float*				bounding
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	extern __shared__ float s_float[];

	float* max_cachex = s_float;
	int cacheIndex = threadIdx.x;
	float temp_maxx = -FLT_MAX;
	while (idx < cloud_size)
	{
		float x = (float)IMG[idx];
		if (temp_maxx < x)
		{
			temp_maxx = x;
		}
		idx += stride;

	}
	max_cachex[cacheIndex] = temp_maxx;
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i)
		{
			if (max_cachex[cacheIndex] < max_cachex[cacheIndex + i])
			{
				max_cachex[cacheIndex] = max_cachex[cacheIndex + i];
			}
		}
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
	{
		bounding[0] = max_cachex[0]+1.0f;
	}
};


int main17()
{
	cv::Mat ima;
	ima = imread("D:\\Stereo3D\\data\\test1209\\BINOCULAR\\phase\\right\\0001.bmp", IMREAD_GRAYSCALE);
	int x_times_y_ = ima.rows*ima.cols;
	float phase_thre_;
	float *d_bounding;
	(hipMalloc((void**)&d_bounding,1 * sizeof(float)));
	unsigned char *image_black_uchar_d_;
	hipMalloc((void**)&image_black_uchar_d_, sizeof(unsigned char)*x_times_y_*(1));
	hipMemcpy(image_black_uchar_d_, ima.data, sizeof(unsigned char)*x_times_y_, hipMemcpyHostToDevice);
	GetThres << <1, 1024, sizeof(float) * 1024 >> > (image_black_uchar_d_, x_times_y_, d_bounding);
	hipDeviceSynchronize();
	hipPeekAtLastError();//3
	hipMemcpy(&phase_thre_, d_bounding, 1 * sizeof(float), hipMemcpyDeviceToHost);
	cout << "phase_thre_:" << phase_thre_ << endl;
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int device;
	hipDeviceProp_t deviceProp;
	for (device = 0; device < deviceCount; ++device)
	{
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d has compute capability %d.%d.\n",
			device, deviceProp.major, deviceProp.minor);
		cout << "regsPerBlock:" << deviceProp.regsPerBlock << endl;
		cout << "maxThreadsPerBlock:" << deviceProp.maxThreadsPerBlock << endl;
		cout << "sharedMemPerBlock:" << deviceProp.sharedMemPerBlock << endl;
	}
	system("PAUSE");
	return 0;
	//hipGetDevice();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int N = 5, M = 5, H = 25, W = 25, D = 2;
	const int PH = H / D + H % D;
	const int image_size = N*M*H*W * sizeof(float);
	const int out_size = N*M*PH*PH * sizeof(float);
	float mul_by = 0.01;
	float *input, *output, *dev_input, *dev_output;


	input = new float[image_size];
	output = new float[out_size];
	for (int i = 0; i<N*M*H*W; i++)
		*(input + i) = i*mul_by;

	hipMalloc((void**)&dev_output, out_size);
	hipMalloc((void**)&dev_input, image_size);
	hipBindTexture(NULL, t_input, dev_input,
		image_size);
	hipMemcpy(dev_input, input, image_size,
		hipMemcpyHostToDevice);

	dim3    grid(M, N);
	dim3    threads(PH, PH);
	hipEventRecord(start, 0);
	MaxPool2d << <grid, threads >> >(H, D, dev_output);
	hipMemcpy(output, dev_output, out_size, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,
		start, stop);
	std::cout << "Time to generate: " << elapsedTime << "ms\n";
	hipEventDestroy(start);
	hipEventDestroy(stop);
	for (int i = 0; i<10; i++)
		std::cout << *(output + i) << std::endl;

	hipFree(dev_input);
	hipFree(dev_output);
	hipUnbindTexture(t_input);
	delete[] output;
	delete[] input;
	system("pause");
}

/*
Time to generate: 0.128448ms
*/