#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
//using namespace cv;
using namespace std;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



#define Row  8
#define Col 4


__global__ void addKernel(int **C, int **A)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;
	if (idx < Col && idy < Row) {
		C[idy][idx] = A[idy][idx] + 10;
	}
}


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void add3DKernelTHREAD(int* a,int *c)
{
    int x = threadIdx.x-1;
    int y = threadIdx.y-1;
    int z = threadIdx.z-1; 
    int iThreadIndex = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;

    c[iThreadIndex * 3] = threadIdx.x - 1;
    c[iThreadIndex * 3 + 1] = threadIdx.y - 1;
    c[iThreadIndex * 3 + 2] = threadIdx.z - 1;
}
__global__ void add3DKernelBLOCK(int* a, int *c)
{
    int x = blockIdx.x - 1;
    int y = blockIdx.y - 1;
    int z = blockIdx.z - 1;
    int iBlockIndex = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    //c[cuurrentindex] = a[cuurr - 1entindex*3]+ a[cuurrentindex * 3+1]*10+ a[cuurrentindex * 3+2]*100;
    //c[cuurrentindex * 3] = a[cuurrentindex * 3];
    //c[cuurrentindex * 3+1] = a[cuurrentindex * 3 + 1];
    //c[cuurrentindex * 3+2] = a[cuurrentindex * 3 + 2];

    c[iBlockIndex * 3] = blockIdx.x - 1;
    c[iBlockIndex * 3 + 1] = blockIdx.y - 1;
    c[iBlockIndex * 3 + 2] = blockIdx.z - 1;
}
__global__ void add3DKernelTHREADBLOCK(int *c)
{
    int iBlockIndex = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int iThreadIndex = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int iCurrentIndex;

    iCurrentIndex = iThreadIndex + iBlockIndex*blockDim.x*blockDim.y*blockDim.z;
    c[iCurrentIndex] = iCurrentIndex;

    
}
__global__ void add3DKernelTHREADBLOCK1(int *c)
{
	int iBlockIndex = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
	int iThreadIndex = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
	int iCurrentIndex;

	iCurrentIndex = iThreadIndex + iBlockIndex*blockDim.x*blockDim.y*blockDim.z;
	c[iCurrentIndex] = iCurrentIndex;


}
__global__ void myTHREADBLOCK(int *d_in,int *d_out,int* size)
{

	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (idx < 10)
	{
		if (d_in[idx] > 0 && d_in[idx] < 6)
		{
			int ind = atomicAdd(size, 1);
			d_out[ind] = d_in[idx];
		}
		
		idx += stride;
	}
}

__global__ void myTHREADBLOCK1(float(*d_GPU)[3], float (*h_GPU)[3], int SIZE)
{

	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (idx < 27)
	{
		
		float *d = (float*)((char*)d_GPU + 512 * idx);
		float *h = (float*)((char*)h_GPU + 512 * idx);
		h[0] = d[0];
		h[1] = d[1];
		h[2] = d[2];
		idx += stride;
	}
}
__global__ void myTHREADBLOCK111(int **d_GPU,int*d)
{

	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	if (idx == 1)
	{
		//int * p = d_GPU[0];
		//int * p1 = d_GPU[1];
		d[0] = 122;
		d[1] = 1222;
		d[2] = d_GPU[1][0];
		d[3] = 1;
		d[4] = 2;
	}
		

}
__global__ void Mytest(float *dev_keys, float *dev_values)
{
	//__shared__ float cache[5];
	////cache[0] = 100.0f;
	//int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;

	//if (idx == 0)
	//{
	//	for (int i = 0; i < 5; i++)
	//	{
	//		dev_keys[i] = cache[i];
	//	}
	//}
	/*return;*/
	/*switch (idx)
	{
	case 1:
		;
		break;
	case 2:
		dev_keys[idx] = __fadd_rn(dev_values[0], dev_values[1]);
		break;
	case 3:
		dev_keys[idx] = __fadd_ru(dev_values[0], dev_values[1]);
		break;
	case 4:
		dev_keys[idx] = __fadd_rz(dev_values[0], dev_values[1]);
		break;
	default:
		break;
	}*/


}

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <iostream>
#include <Windows.h>
#define gpuErrchk(severity_level, ans) \
    if (ans != hipSuccess)\
    {\
        std::string func_info = FUNCTION_INFO; \
        hipError_t err_code = ans;\
        throw ExceptionsS3D(hipGetErrorString(err_code), func_info);\
    }
//int main(void)
//{
//	LARGE_INTEGER m_liPerfFreq = { 0 };
//	//��ȡÿ�����CPU Performance Tick 
//	QueryPerformanceFrequency(&m_liPerfFreq);
//	LARGE_INTEGER m_liPerfStart = { 0 };
//	QueryPerformanceCounter(&m_liPerfStart);
//	LARGE_INTEGER m_liPerfEnd = { 0 };
//	int point_cloud_size = 8000000;
//	int hashvaluearray_size = 27 * point_cloud_size;
//	int *d_hashtable;//ÿ����Ĺ�ϣֵ
//	hipMalloc((void**)&d_hashtable, point_cloud_size * sizeof(int));
//	int *d_hashtableold;//ÿ����Ĺ�ϣֵ
//	hipMalloc((void**)&d_hashtableold, point_cloud_size * sizeof(int));
//	int *d_hashindex;
//	hipMalloc((void**)&d_hashindex, point_cloud_size * sizeof(int));
//	int* d_hashvaluearray;
//	hipMalloc((void**)&d_hashvaluearray, hashvaluearray_size * sizeof(int));
//	// ����CPU���е����ڵ�ʱ��
//	QueryPerformanceCounter(&m_liPerfEnd);
//	int time = (((m_liPerfEnd.QuadPart - m_liPerfStart.QuadPart) * 1000) / m_liPerfFreq.QuadPart);//ms
//	printf("Pre Time Slaps:%d ms\n", time);
//	//����hashֵ-����-��ȡ��С����ʼֵ-������ϣֵ�б�-ǰ����-ȥ��
//
//	QueryPerformanceCounter(&m_liPerfStart);
//	hipFree(d_hashtable);
//	hipFree(d_hashindex);
//	hipFree(d_hashtableold);
//	hipFree(d_hashvaluearray);
//	QueryPerformanceCounter(&m_liPerfEnd);
//	time = (((m_liPerfEnd.QuadPart - m_liPerfStart.QuadPart) * 1000) / m_liPerfFreq.QuadPart);//ms
//	printf("Release Resource Time Slaps:%d ms\n", time);
//
//	QueryPerformanceCounter(&m_liPerfStart);
//	int *d_data;//ÿ����Ĺ�ϣֵ
//	hipMalloc((void**)&d_data, 30*point_cloud_size * sizeof(int));
//	// ����CPU���е����ڵ�ʱ��
//	QueryPerformanceCounter(&m_liPerfEnd);
//	time = (((m_liPerfEnd.QuadPart - m_liPerfStart.QuadPart) * 1000) / m_liPerfFreq.QuadPart);//ms
//	printf("Pre Time Slaps:%d ms\n", time);
//
//	QueryPerformanceCounter(&m_liPerfStart);
//	hipFree(d_data);
//	QueryPerformanceCounter(&m_liPerfEnd);
//	time = (((m_liPerfEnd.QuadPart - m_liPerfStart.QuadPart) * 1000) / m_liPerfFreq.QuadPart);//ms
//	printf("Release Resource Time Slaps:%d ms\n", time);
//	system("PAUSE");
//	return 0;
//}

#define getTimeGeneral(clock_t_)  (clock_t_)*1.0/ CLOCKS_PER_SEC * 1000
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
int main111()
{
	//int deviceCount;
	//hipGetDeviceCount(&deviceCount);
	//int device;
	//hipDeviceProp_t deviceProp;
	//for (device = 0; device < deviceCount; ++device)
	//{
	//	hipGetDeviceProperties(&deviceProp, device);
	//	printf("Device %d has compute capability %d.%d.\n",
	//		device, deviceProp.major, deviceProp.minor);
	//	cout << "regsPerBlock:" << deviceProp.regsPerBlock << endl;
	//	cout << "maxThreadsPerBlock:" << deviceProp.maxThreadsPerBlock << endl;
	//	cout << "sharedMemPerBlock:" << deviceProp.sharedMemPerBlock << endl;
	//}
	//system("PAUSE");
	//return 0;
	const int N = 5;
	float values[N] = { 1.5, 1.1, 1.1, 1.1, 1.1 };
	float keys[N] = { 0 };

	//thrust::sort_by_key(keys, keys + N, values);
	clock_t a, b;
	float * h_ = new float[50000000];
	float *dev_ = 0;
	float *dev_1 = 0;
	int count = 1000000;
	hipMalloc((void**)&dev_, 50000000 * sizeof(float));
	hipMemcpy(dev_, h_, 5000000 * sizeof(float), hipMemcpyHostToDevice);
	// Allocate GPU buffers for three vectors (two input, one output)    .
	for (int i = 0; i < 10; i++)
	{
		std::cout<< std::endl << std::endl;
		{
			a = clock();
			for (int i = 0; i<10; i++)
			{
				hipMemcpy(dev_, h_, 5000000 * sizeof(float), hipMemcpyHostToDevice);
			}
			hipError_t error = hipDeviceSynchronize();
			error = hipPeekAtLastError();
			b = clock();
			std::cout << "reconstruct time of 100 times 1:" << getTimeGeneral(b - a) << std::endl;
		}
		{
			a = clock();
			hipMemcpy(dev_, h_, 50000000 * sizeof(float), hipMemcpyHostToDevice);
			hipError_t error = hipDeviceSynchronize();
			error = hipPeekAtLastError();
			b = clock();
			std::cout << "reconstruct time of 1 times 100:" << getTimeGeneral(b - a) << std::endl;
		}
	} 
		delete[]h_;
		hipFree(dev_);
	//float *dev_keys = 0;
	//float *dev_values = 0;
	//float dev_c[5] = {1,2,3,4,5};
	//// Allocate GPU buffers for three vectors (two input, one output)    .
	//hipMalloc((void**)&dev_keys, N * sizeof(float));
	//hipMalloc((void**)&dev_values, N * sizeof(float));


	//hipMemcpy(dev_keys, keys, N * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_values, values, N * sizeof(float), hipMemcpyHostToDevice);

	//Mytest << < 65535, 1024 >> >(dev_keys, dev_values);
	//hipError_t error= hipDeviceSynchronize();
	//error = hipPeekAtLastError();
	//////hipMemcpy(dev_c, dev_keys, N * sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}
#include <opencv2/opencv.hpp>
namespace GPU = cv::cuda;

__global__ void kernel(float *src,int rows,int cols,int channel)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int size = cols*rows;
	while (idx < size*channel)
	{
		//src(distorted_y,distorted_x) = 10*distorted_y+ distorted_x;
		src[idx] = 9.96;
		idx += stride;
	}
}

//int main() 
//{
//	// ����Ҫ����Ƿ�CUDAģ���Ƿ����
//	//if (GPU::getCudaEnabledDeviceCount() == 0) {
//	//	cerr << "��OpenCV�����ʱ��û������CUDAģ��" << endl;
//	//	return -1;
//	//}
//	int aaa[8] = { 0 };
//	int  *d_hashtable1;//ÿ����Ĺ�ϣֵ
//	hipMalloc((void**)&d_hashtable1, 8 * sizeof(int));
//	hipMemset(d_hashtable1, 1, 8 * sizeof(int));
//	hipMemcpy(aaa, d_hashtable1, 8 * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemset(d_hashtable1, 255, 8 * sizeof(int));
//	float a1[8] = { 1,2,3,4,5,6,7,8 };
//	float *b = new float[8];
//	memcpy(b, a1, 32);
//	Mat temp20 = Mat(Size(4, 2), CV_32F, b);
//	delete[]b;
//	return 0;
//	Mat temp = Mat(2, 3, CV_32FC2);
//	const int rows = 16 * 50;
//	const int cols = 32 * 60;
//	const int type = CV_8UC3;
//	Mat one = Mat::ones(2, 2, CV_8UC2);
//	float a[8] = { 1,2,3,4,5,6,7,8 };
//	Mat temp2= Mat(Size(4,2), CV_32F,a);
//	Mat temp3 = Mat(2,4, CV_32F, a);
//	a[7] = 10000;
//	float *d_hashtable;//ÿ����Ĺ�ϣֵ
//	hipMalloc((void**)&d_hashtable, 8 * sizeof(float));
//	hipMemcpy(d_hashtable, a, 8 * sizeof(float), hipMemcpyHostToDevice);
//	GPU::GpuMat gpuMat1 = GPU::GpuMat(2, 4, CV_32FC1, d_hashtable);
//	gpuMat1.download(temp);
//	hipMemset(d_hashtable, 1, 4 * 4);
//	gpuMat1.download(temp);
//	cuda::subtract(GPU::GpuMat(2, 4, CV_32FC1, d_hashtable), GPU::GpuMat(2, 4, CV_32FC1, d_hashtable), gpuMat1);
//	// ��ʼ��һ����ɫ��GpuMat
//	/*GPU::GpuMat gpuMat(2,4, CV_32FC1, d_hashtable);*/
//	gpuMat1.download(temp);
//	hipMemset(d_hashtable, 1, 4 * 4);
//	gpuMat1.download(temp);
//	hipMemcpy(a, d_hashtable, 8 * sizeof(float), hipMemcpyDeviceToHost);
//	Mat temp1 = Mat(2, 4, CV_32S);
//	memcpy(temp1.data,a,8*4);
//	temp1.convertTo(temp1, CV_32F);
//	GPU::GpuMat gpuMat;/*= GPU::GpuMat(2, 4, CV_32FC1, Scalar(0))*/;
//	gpuMat.upload(temp1);
//	//{
//	//	gpuMat = GPU::GpuMat(100, 1000, CV_32FC1, Scalar(0));
//	//}
//	//gpuMat(0, 0) = 100;
//	//hipFree(d_hashtable);
//	//vector<Mat> abc;
//	Mat two;
//	//GPU::GpuMat gpuMat1;
//	//
//	////Mat temp/* = Mat(ROWS, COLS, CV_32FC2)*/;
//	////merge(&gpuMat[0], 2, temp);
//	//gpuMat.convertTo(gpuMat1, CV_8UC2);
//	//gpuMat.download(two);
//	//gpuMat1.download(two);
//	//kernel << <100, 100 >> >((float*)gpuMat.data,2,2,2);
//	
//	gpuMat.download(two);
//	return 0;
//}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
